#include "hip/hip_runtime.h"
#include <ATen/cuda/Atomic.cuh>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: points(b, c, n) idx(b, m)
// output: out(b, c, m)
template <typename scalar_t>
__global__ void gather_points_kernel(int b, int c, int n, int m,
                                     const scalar_t *__restrict__ points,
                                     const int *__restrict__ idx,
                                     scalar_t *__restrict__ out) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int l = blockIdx.y; l < c; l += gridDim.y) {
      for (int j = threadIdx.x; j < m; j += blockDim.x) {
        int a = idx[i * m + j];
        out[(i * c + l) * m + j] = points[(i * c + l) * n + a];
      }
    }
  }
}

void gather_points_kernel_wrapper(int b, int c, int n, int npoints,
                                  const float *points, const int *idx,
                                  float *out) {
  gather_points_kernel<float><<<dim3(b, c, 1), opt_n_threads(npoints), 0,
                                at::cuda::getCurrentCUDAStream()>>>(
      b, c, n, npoints, points, idx, out);

  CUDA_CHECK_ERRORS();
}

void gather_points_kernel_wrapper_bf16(int b, int c, int n, int npoints,
                                       const at::BFloat16 *points,
                                       const int *idx, at::BFloat16 *out) {
  gather_points_kernel<at::BFloat16><<<dim3(b, c, 1), opt_n_threads(npoints), 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
      b, c, n, npoints, points, idx, out);

  CUDA_CHECK_ERRORS();
}

// input: grad_out(b, c, m) idx(b, m)
// output: grad_points(b, c, n)
template <typename scalar_t>
__global__ void gather_points_grad_kernel(int b, int c, int n, int m,
                                          const scalar_t *__restrict__ grad_out,
                                          const int *__restrict__ idx,
                                          scalar_t *__restrict__ grad_points) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int l = blockIdx.y; l < c; l += gridDim.y) {
      for (int j = threadIdx.x; j < m; j += blockDim.x) {
        int a = idx[i * m + j];
        gpuAtomicAdd(grad_points + (i * c + l) * n + a,
                     grad_out[(i * c + l) * m + j]);
      }
    }
  }
}

void gather_points_grad_kernel_wrapper(int b, int c, int n, int npoints,
                                       const float *grad_out, const int *idx,
                                       float *grad_points) {
  gather_points_grad_kernel<float><<<dim3(b, c, 1), opt_n_threads(npoints), 0,
                                     at::cuda::getCurrentCUDAStream()>>>(
      b, c, n, npoints, grad_out, idx, grad_points);

  CUDA_CHECK_ERRORS();
}

void gather_points_grad_kernel_wrapper_bf16(int b, int c, int n, int npoints,
                                            const at::BFloat16 *grad_out,
                                            const int *idx,
                                            at::BFloat16 *grad_points) {
  gather_points_grad_kernel<at::BFloat16>
      <<<dim3(b, c, 1), opt_n_threads(npoints), 0,
         at::cuda::getCurrentCUDAStream()>>>(b, c, n, npoints, grad_out, idx,
                                             grad_points);

  CUDA_CHECK_ERRORS();
}

template <typename scalar_t>
__device__ void __update(scalar_t *__restrict__ dists,
                         int *__restrict__ dists_i, int idx1, int idx2) {
  const scalar_t v1 = dists[idx1], v2 = dists[idx2];
  const int i1 = dists_i[idx1], i2 = dists_i[idx2];
  dists[idx1] = v2 > v1 ? v2 : v1;
  dists_i[idx1] = v2 > v1 ? i2 : i1;
}

// Input dataset: (b, n, 3), tmp: (b, n)
// Ouput idxs (b, m)
template <typename scalar_t, unsigned int block_size>
__global__ void furthest_point_sampling_kernel(
    int b, int n, int m, const scalar_t *__restrict__ dataset,
    scalar_t *__restrict__ temp, int *__restrict__ idxs) {
  if (m <= 0)
    return;
  __shared__ scalar_t dists[block_size];
  __shared__ int dists_i[block_size];

  int batch_index = blockIdx.x;
  dataset += batch_index * n * 3;
  temp += batch_index * n;
  idxs += batch_index * m;

  int tid = threadIdx.x;
  const int stride = block_size;

  int old = 0;
  if (threadIdx.x == 0)
    idxs[0] = old;

  __syncthreads();
  for (int j = 1; j < m; j++) {
    int besti = 0;
    scalar_t best = static_cast<scalar_t>(-1);
    scalar_t x1 = dataset[old * 3 + 0];
    scalar_t y1 = dataset[old * 3 + 1];
    scalar_t z1 = dataset[old * 3 + 2];
    for (int k = tid; k < n; k += stride) {
      scalar_t x2, y2, z2;
      x2 = dataset[k * 3 + 0];
      y2 = dataset[k * 3 + 1];
      z2 = dataset[k * 3 + 2];
      scalar_t mag = (x2 * x2) + (y2 * y2) + (z2 * z2);
      if (mag <= static_cast<scalar_t>(1e-3))
        continue;

      scalar_t d =
          (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);

      scalar_t d2 = d < temp[k] ? d : temp[k];
      temp[k] = d2;
      besti = d2 > best ? k : besti;
      best = d2 > best ? d2 : best;
    }
    dists[tid] = best;
    dists_i[tid] = besti;
    __syncthreads();

    if (block_size >= 512) {
      if (tid < 256) {
        __update<scalar_t>(dists, dists_i, tid, tid + 256);
      }
      __syncthreads();
    }
    if (block_size >= 256) {
      if (tid < 128) {
        __update<scalar_t>(dists, dists_i, tid, tid + 128);
      }
      __syncthreads();
    }
    if (block_size >= 128) {
      if (tid < 64) {
        __update<scalar_t>(dists, dists_i, tid, tid + 64);
      }
      __syncthreads();
    }
    if (block_size >= 64) {
      if (tid < 32) {
        __update<scalar_t>(dists, dists_i, tid, tid + 32);
      }
      __syncthreads();
    }
    if (block_size >= 32) {
      if (tid < 16) {
        __update<scalar_t>(dists, dists_i, tid, tid + 16);
      }
      __syncthreads();
    }
    if (block_size >= 16) {
      if (tid < 8) {
        __update<scalar_t>(dists, dists_i, tid, tid + 8);
      }
      __syncthreads();
    }
    if (block_size >= 8) {
      if (tid < 4) {
        __update<scalar_t>(dists, dists_i, tid, tid + 4);
      }
      __syncthreads();
    }
    if (block_size >= 4) {
      if (tid < 2) {
        __update<scalar_t>(dists, dists_i, tid, tid + 2);
      }
      __syncthreads();
    }
    if (block_size >= 2) {
      if (tid < 1) {
        __update<scalar_t>(dists, dists_i, tid, tid + 1);
      }
      __syncthreads();
    }

    old = dists_i[0];
    if (tid == 0)
      idxs[j] = old;
  }
}

void furthest_point_sampling_kernel_wrapper(int b, int n, int m,
                                            const float *dataset, float *temp,
                                            int *idxs) {
  unsigned int n_threads = opt_n_threads(n);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (n_threads) {
  case 512:
    furthest_point_sampling_kernel<float, 512>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 256:
    furthest_point_sampling_kernel<float, 256>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 128:
    furthest_point_sampling_kernel<float, 128>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 64:
    furthest_point_sampling_kernel<float, 64>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 32:
    furthest_point_sampling_kernel<float, 32>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 16:
    furthest_point_sampling_kernel<float, 16>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 8:
    furthest_point_sampling_kernel<float, 8>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 4:
    furthest_point_sampling_kernel<float, 4>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 2:
    furthest_point_sampling_kernel<float, 2>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 1:
    furthest_point_sampling_kernel<float, 1>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  default:
    furthest_point_sampling_kernel<float, 512>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
  }

  CUDA_CHECK_ERRORS();
}

void furthest_point_sampling_kernel_wrapper_bf16(int b, int n, int m,
                                                 const at::BFloat16 *dataset,
                                                 at::BFloat16 *temp,
                                                 int *idxs) {
  unsigned int n_threads = opt_n_threads(n);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (n_threads) {
  case 512:
    furthest_point_sampling_kernel<at::BFloat16, 512>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 256:
    furthest_point_sampling_kernel<at::BFloat16, 256>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 128:
    furthest_point_sampling_kernel<at::BFloat16, 128>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 64:
    furthest_point_sampling_kernel<at::BFloat16, 64>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 32:
    furthest_point_sampling_kernel<at::BFloat16, 32>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 16:
    furthest_point_sampling_kernel<at::BFloat16, 16>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 8:
    furthest_point_sampling_kernel<at::BFloat16, 8>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 4:
    furthest_point_sampling_kernel<at::BFloat16, 4>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 2:
    furthest_point_sampling_kernel<at::BFloat16, 2>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  case 1:
    furthest_point_sampling_kernel<at::BFloat16, 1>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    break;
  default:
    furthest_point_sampling_kernel<at::BFloat16, 512>
        <<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
  }

  CUDA_CHECK_ERRORS();
}
