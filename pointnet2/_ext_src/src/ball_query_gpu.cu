#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
template <typename scalar_t>
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int nsample,
                                        const scalar_t *__restrict__ new_xyz,
                                        const scalar_t *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    scalar_t new_x = new_xyz[j * 3 + 0];
    scalar_t new_y = new_xyz[j * 3 + 1];
    scalar_t new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      scalar_t x = xyz[k * 3 + 0];
      scalar_t y = xyz[k * 3 + 1];
      scalar_t z = xyz[k * 3 + 2];
      scalar_t d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                    (new_z - z) * (new_z - z);
      if (static_cast<float>(d2) < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_point_kernel_wrapper(int b, int n, int m, float radius,
                                     int nsample, const float *new_xyz,
                                     const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel<float><<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}

void query_ball_point_kernel_wrapper_bf16(int b, int n, int m, float radius,
                                          int nsample,
                                          const at::BFloat16 *new_xyz,
                                          const at::BFloat16 *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel<at::BFloat16><<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}
